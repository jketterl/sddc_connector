#include "hip/hip_runtime.h"
#include "sdr_cuda.hpp"
#include <hip/hip_fp16.h>
#include <iostream>

using namespace SdrCuda;

__global__ void convert_ui16_c_kernel(int16_t* input, float* output, double* phase_offset, double* angle_per_sample) {
    uint32_t i = blockDim.x * blockIdx.x + threadIdx.x;
    float converted = (float)input[i] / INT16_MAX;
    double angle = *phase_offset + *angle_per_sample * i;
    double sinValue;
    double cosValue;
    sincospi(angle, &sinValue, &cosValue);
    output[i * 2] = sinValue * converted;
    output[i * 2 + 1] = cosValue * converted;
}

__global__ void fir_decimate_c_kernel(float* input, float* output, uint16_t* decimation, uint16_t* decimation_offset, float* taps, uint16_t taps_length) {
    uint32_t i = blockDim.x * blockIdx.x + threadIdx.x;
    uint32_t dec_i = *decimation_offset + i * *decimation;
    float acci = 0;
    float accq = 0;
    for (uint16_t k = 0; k < taps_length; k++) {
        int32_t index = dec_i - (taps_length - k);
        acci += input[index * 2] * taps[k];
        accq += input[index * 2 + 1] * taps[k];
    }
    output[i * 2] = acci;
    output[i * 2 + 1] = accq;
}

__global__ void fir_decimate_copy_delay(float* input, float* output) {
    uint32_t i = blockDim.x * blockIdx.x + threadIdx.x;
    output[i * 2] = input[i * 2];
    output[i * 2 + 1] = input[i * 2 + 1];
}

Ddc::Ddc(uint32_t new_buffersize) {
    buffersize = new_buffersize;

    hipMalloc((void**)&decimation_device, sizeof(uint16_t));
    hipMemcpy(decimation_device, &decimation, sizeof(uint16_t), hipMemcpyHostToDevice);

    hipMalloc((void**)&decimation_offset_device, sizeof(uint16_t));
    hipMemcpy(decimation_offset_device, &decimation_offset, sizeof(uint16_t), hipMemcpyHostToDevice);

    hipMalloc((void**)&phase_offset_device, sizeof(double));
    hipMemcpy(phase_offset_device, &phase_offset, sizeof(double), hipMemcpyHostToDevice);

    hipMalloc((void**)&angle_per_sample_device, sizeof(double));

    hipMalloc((void**)&raw, sizeof(int16_t) * buffersize);
    hipMalloc((void**)&input, sizeof(float) * 2 * (buffersize + taps_length));
    hipMalloc((void**)&output, sizeof(float) * buffersize);

    reconfigure();
}

void Ddc::set_frequency_offset(float new_frequency_offset) {
    freq_offset = new_frequency_offset;
    reconfigure();
}

void Ddc::set_decimation(uint16_t new_decimation) {
    decimation = new_decimation;
    hipMemcpy(decimation_device, &decimation, sizeof(uint16_t), hipMemcpyHostToDevice);

    decimation_offset = 0;
    hipMemcpy(decimation_offset_device, &decimation_offset, sizeof(uint16_t), hipMemcpyHostToDevice);
    reconfigure();
}

void Ddc::reconfigure() {
    taps_length = 4 * decimation;
    if (taps_length %2 == 0) taps_length++;
    // maximum number of taps currently limited by maximum number of cuda threads
    if (taps_length > 511) taps_length = 511;
    taps_length = max(taps_length, 121);
    std::cerr << "taps length: " << taps_length << "\n";

    float* new_taps = (float*) malloc(sizeof(float) * taps_length);
    firdes_lowpass_f(new_taps, taps_length, 0.485/decimation, WINDOW_HAMMING);

    if (taps != nullptr) hipFree(taps);
    hipMalloc((void**)&taps, sizeof(float) * taps_length);
    hipMemcpy(taps, new_taps, sizeof(float) * taps_length, hipMemcpyHostToDevice);

    free(new_taps);

    angle_per_sample = 2 * freq_offset;
    hipMemcpy(angle_per_sample_device, &angle_per_sample, sizeof(double), hipMemcpyHostToDevice);
}

Ddc::~Ddc() {
    hipFree(decimation_device);
    hipFree(decimation_offset_device);
    hipFree(phase_offset_device);
    hipFree(angle_per_sample_device);
    hipFree(raw);
    hipFree(input);
    hipFree(output);
    hipFree(taps);
}

float* Ddc::alloc_output(uint32_t length) {
    float* output;
    hipHostAlloc((void**)&output, sizeof(float) * 2 * length, hipHostMallocDefault);
    return output;
}

void Ddc::free_output(float* output) {
    hipFree(output);
}

float* Ddc::get_fir_decimate_input() {
    return input + (taps_length * 2);
}

uint32_t Ddc::run(int16_t* input_samples, float* host_output, uint32_t length) {
    hipMemcpy(raw, input_samples, sizeof(int16_t) * length, hipMemcpyHostToDevice);

    int blocks = length / 1024;
    // run an extra block if memory does not line up ideally
    if (blocks % 1024 > 0) blocks += 1;
    convert_ui16_c_kernel<<<blocks, 1024>>>(raw, get_fir_decimate_input(), phase_offset_device, angle_per_sample_device);

    // move the phase forward
    phase_offset += angle_per_sample * length;
    while (phase_offset >= 2) phase_offset -= 2;
    hipMemcpy(phase_offset_device, &phase_offset, sizeof(double), hipMemcpyHostToDevice);

    uint32_t out_samples = (decimation_offset + length) / decimation;
    blocks = out_samples / 512;
    // run an extra block if memory does not line up ideally
    if (out_samples % 512 > 0) blocks += 1;
    fir_decimate_c_kernel<<<blocks, 512>>>(get_fir_decimate_input(), output, decimation_device, decimation_offset_device, taps, taps_length);

    // update decimation offset
    decimation_offset = (decimation_offset + length) % decimation;
    hipMemcpy(decimation_offset_device, &decimation_offset, sizeof(uint16_t), hipMemcpyHostToDevice);

    // copy unprocessed samples from the end to the beginning of the input buffer
    fir_decimate_copy_delay<<<1, taps_length>>>(input + (length * 2), input);
    hipMemcpy(host_output, output, sizeof(float) * (out_samples * 2), hipMemcpyDeviceToHost);

    hipDeviceSynchronize();
    hipError_t error = hipGetLastError();
    if (error != hipSuccess) {
        std::cerr << "CUDA ERROR: " << hipGetErrorString(error) << "\n";
        exit(-1);
    }

    return out_samples;
}
